#include "hip/hip_runtime.h"
/* 3D elastic time-domain FD modeling with multiple GPUs coordinated via MPI and p2p*/

/*
  Authors: Robin M. Weiss and Jeffrey Shragge

  This code is a GPU-enabled version of the ewefd3d module from the Madagascar
  software package (see: http://www.reproducibility.org).  It implements a 3D
  Finite-Difference Time Domain solver for the elastice wave equation with 
  2nd- and 8th- order temporal and spatial accuracy, respectively.  Computation
  is distributed across an arbitrary number of GPU devices and coordinted by MPI.
  Inter-GPU communication that occurs within a node is carried out with direct p2p
  communication over the PCIe bus via cudaMemCpy(DeviceToDevice).  Communication
  between GPUs located in seperate nodes occurs via MPI send/receive calls. 
  For more information, see (Weiss and Shragge, "Solving 3D Anisotropic Elastic Wave 
  Equations on Parallel GPU Devices", GEOPHYSICS. http://software.seg.org/2012/0063)
*/

/*
  Copyright (C) 2012 University of Western Australia
  
  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.
  
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/

/*

SOMETHING YOU SHOULD PROBABLY KNOW ABOUT THIS CODE:

All arrays used internally in this program order axes Y, Z, X (slowest to fastest)
However, madagascar wants axes to be ordered Y, X, Z and this is the standard in RSF files (i think)

Therefore, whenever reading arrays from a file, or writing data to a file, the X and Z axes must be transposed.

I appologize for the inconvenience and encourage someone to take the time to fix this and make the internal representation the same as the external.

sorry for being difficult,
---Robin

*/



#include <stdio.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <rsf.h>
}

#include "fdutil.c"
#include "ewefd3d_multiNode_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define NOP 4 /* derivative operator half-size */


// checks the current GPU device for an error flag and prints to stderr
static void sf_check_gpu_error (int rank, const char *msg) {
    hipError_t err = hipGetLastError ();
     if (hipSuccess != err)
        sf_error ("Cuda error on node %d: %s: %s", rank, msg, hipGetErrorString (err));
}




// entry point
int main (int argc, char* argv[]) {
	
	// Initialize MPI
	int rank, nNode;
	MPI_Init (&argc, &argv);				/* start MPI */
	MPI_Comm_rank (MPI_COMM_WORLD, &rank);	/* get current process id */
	MPI_Comm_size (MPI_COMM_WORLD, &nNode);	/* get number of nodes */
	MPI_Status status;
	MPI_Request request;
	
	MPI_Request *recvRequests = (MPI_Request*)malloc(12 * sizeof(MPI_Request));
	MPI_Status *recvStatuses = (MPI_Status*)malloc(12 * sizeof(MPI_Status));
	
	// Initialize RSF
	sf_init(argc, argv);

	bool verb,fsrf,snap,ssou,dabc,interp,wavSrc;
    int  jsnap,ntsnap,jdata;

    /* I/O files */
    sf_file Fwav=NULL; /* wavelet   */
    sf_file Fsou=NULL; /* sources   */
    sf_file Frec=NULL; /* receivers */
    sf_file Fccc=NULL; /* velocity  */
    sf_file Fden=NULL; /* density   */
    sf_file Fdat=NULL; /* data      */
    sf_file Fwfl=NULL; /* wavefield */

	sf_file Fum=NULL;
	sf_file Fuo=NULL;

	/* cube axes */
    sf_axis at,ax,ay,az;
    sf_axis as,ar,ac;

    int     nt,nz,nx,ny,ns,nr,nc,nb;
    int     it,iz,ix,iy;
    float   dt,dz,dx,dy,idz,idx,idy;

    /* FDM structure */
    fdm3d    fdm=NULL;

    /* I/O arrays */
    float***ww=NULL;           /* wavelet   */
    pt3d   *ss=NULL;           /* sources   */
    pt3d   *rr=NULL;           /* receivers */

	
    /*------------------------------------------------------------*/
    /* displacement: um = U @ t-1; uo = U @ t; up = U @ t+1 */

	// used for writing wavefield to file, only needed if snap=y
	float ***uox, ***uoy, ***uoz;
	float *h_uox, *h_uoy, *h_uoz;

    /*------------------------------------------------------------*/
    /* linear interpolation weights/indices */
    lint3d cs,cr;

    /* Gaussian bell */
    int nbell;
    
    /* wavefield cut params */
    sf_axis   acz=NULL,acx=NULL,acy=NULL;
    int       nqz,nqx,nqy;
    float     oqz,oqx,oqy;
    float     dqz,dqx,dqy;
    float     ***uc=NULL;

	/*------------------------------------------------------------*/
    /* execution flags */
    if(! sf_getbool("verb",&verb)) verb=false; /* verbosity flag */
    if(! sf_getbool("snap",&snap)) snap=false; /* wavefield snapshots flag */
    if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
    if(! sf_getbool("ssou",&ssou)) ssou=false; /* stress source */
    if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
	if(! sf_getbool("interp",&interp)) interp=true; /* perform linear interpolation on receiver locations */
	if(! sf_getbool("wavSrc",&wavSrc)) wavSrc=true; /* if yes, look for a source wavelet.  if no, look for initial displacement fields (uo and um) */
    /*------------------------------------------------------------*/


    /*------------------------------------------------------------*/
    /* I/O files */
    Fwav = sf_input ("wav"); /* source wavelet.  must always be present to establish time but contents is ignored if wavSrc=n   */
    Fccc = sf_input ("ccc"); /* stiffness */
    Fden = sf_input ("den"); /* density   */
	Fsou = sf_input ("sou"); /* source locations   */
    Frec = sf_input ("rec"); /* receiver locations */
    Fwfl = sf_output("wfl"); /* wavefield */
    Fdat = sf_output("out"); /* data      */
    
	if (!wavSrc){
		Fum  = sf_input ("um"); /* if wavSrc=n, looks for um file containing the previous displacement timestep. axes: z, x, then y */
		Fuo  = sf_input ("uo"); /* if wavSrc=n, looks for uo file containing the current displacement timestep. axes: z, x, then y */
	}
	/*------------------------------------------------------------*/
	
	/*------------------------------------------------------------*/
    /* initialize GPU */

	int availGPU;
	hipGetDeviceCount(&availGPU);
	
	int ngpu;
	if(!sf_getint("ngpu",&ngpu)) ngpu=1; 	/* Number of GPUs in each node, must be set to lowest common number of GPUs*/

	if (availGPU < ngpu){
		sf_error("ngpu parameter (%d) is greater than %d GPUs in node %d", ngpu, availGPU, rank);
	}
	
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	}
	/*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
    /* axes */
    at = sf_iaxa(Fwav,3); sf_setlabel(at,"t"); if(verb && rank == 0) sf_raxa(at); /* time */
    az = sf_iaxa(Fccc,1); sf_setlabel(az,"z"); if(verb && rank == 0) sf_raxa(az); /* depth */
    ax = sf_iaxa(Fccc,2); sf_setlabel(ax,"x"); if(verb && rank == 0) sf_raxa(ax); /* space x */
    ay = sf_iaxa(Fccc,3); sf_setlabel(ay,"y"); if(verb && rank == 0) sf_raxa(ay); /* space y */

    as = sf_iaxa(Fsou,2); sf_setlabel(as,"s"); if(verb && rank == 0) sf_raxa(as); /* sources */
    ar = sf_iaxa(Frec,2); sf_setlabel(ar,"r"); if(verb && rank == 0) sf_raxa(ar); /* receivers */

    nt = sf_n(at); dt = sf_d(at);
    nz = sf_n(az); dz = sf_d(az);
    nx = sf_n(ax); dx = sf_d(ax);
    ny = sf_n(ay); dy = sf_d(ay);

    ns = sf_n(as);
    nr = sf_n(ar);
    /*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
    /* other execution parameters */
    if(! sf_getint("nbell",&nbell)) nbell=5;  /* bell size */
    if(verb && rank == 0) sf_warning("nbell=%d",nbell);
    if(! sf_getint("jdata",&jdata)) jdata=1;	/* extract receiver data every jdata time steps */
    if(snap) {  
	if(! sf_getint("jsnap",&jsnap)) jsnap=nt; /* save wavefield every jsnap time steps */
    }
    /*------------------------------------------------------------*/


    /*------------------------------------------------------------*/
    /* expand domain for FD operators and ABC */
    if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;

	fdm=fdutil3d_init(verb,fsrf,az,ax,ay,nb,1);
	
	sf_setn(az,fdm->nzpad); sf_seto(az,fdm->ozpad); if(verb && rank == 0) sf_raxa(az);
    sf_setn(ax,fdm->nxpad); sf_seto(ax,fdm->oxpad); if(verb && rank == 0) sf_raxa(ax);
    sf_setn(ay,fdm->nypad); sf_seto(ay,fdm->oypad); if(verb && rank == 0) sf_raxa(ay);
	/*------------------------------------------------------------*/
	
	
	/*------------------------------------------------------------*/
    /* check dimensions of all other files */
	if (sf_n(sf_iaxa(Fden,1)) != nz) sf_error("Dimension missmatch on z-axis of density model");
	if (sf_n(sf_iaxa(Fden,2)) != nx) sf_error("Dimension missmatch on x-axis of density model");
	if (sf_n(sf_iaxa(Fden,3)) != ny) sf_error("Dimension missmatch on y-axis of density model");
	
	if (!wavSrc){
		if (sf_n(sf_iaxa(Fuo,1)) != nz+2*fdm->nb) sf_error("Dimension missmatch on z-axis of initial displacement field (uo)");
		if (sf_n(sf_iaxa(Fuo,2)) != nx+2*fdm->nb) sf_error("Dimension missmatch on x-axis of initial displacement field (uo)");
		if (sf_n(sf_iaxa(Fuo,3)) != ny+2*fdm->nb) sf_error("Dimension missmatch on y-axis of initial displacement field (uo)");
		
		if (sf_n(sf_iaxa(Fum,1)) != nz+2*fdm->nb) sf_error("Dimension missmatch on z-axis of initial displacement field (um)");
		if (sf_n(sf_iaxa(Fum,2)) != nx+2*fdm->nb) sf_error("Dimension missmatch on x-axis of initial displacement field (um)");
		if (sf_n(sf_iaxa(Fum,3)) != ny+2*fdm->nb) sf_error("Dimension missmatch on y-axis of initial displacement field (um)");
	}
	
	
	/*------------------------------------------------------------*/
    /* compute sub-domain dimmensions (domain decomposition) */


	// check that dimmeionsons are ok for FD kernels
	if ((fdm->nzpad - 8) % 24 != 0){
		sf_error("nz + 2*nb - 8 is not a multiple of 24");
	}
	if ((fdm->nxpad - 8) % 24 != 0){
		sf_error("nx + 2*nb - 8 is not a multiple of 24");
	}
	if ((fdm->nypad % (ngpu * nNode)) != 0){
		sf_error("You are using %d GPUs.\n(ny + 2*nb) must me a multiple of %d\nChange model dimensions or select a different number of GPUs", ngpu, ngpu);
	}
	
	
	int nyinterior = (fdm->nypad / (nNode * ngpu) ); // size of sub-domains in y-dimension EXCLUDING any ghost cells from adjacent GPUs
	
	// nodes require 8 additional ghost slices
	int nylocal = nyinterior + 8;
	
	// if using 1 GPU, this GPU holds the entire domain
	if (ngpu == 1 && nNode == 1){
		nylocal = fdm->nypad;
	}
	
	/*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
    /* setup bell for source injection smoothing */
	if (nbell * 2 + 1 > 32){
		sf_error("nbell must be <= 15\n");
	}
	
	float *h_bell;
	h_bell = (float*)malloc((2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float));
	
	float s = 0.5*nbell;
    for (iy=-nbell;iy<=nbell;iy++) {
		for (ix=-nbell;ix<=nbell;ix++) {
	    	for(iz=-nbell;iz<=nbell;iz++) {
				h_bell[(iy + nbell) * (2*nbell+1) * (2*nbell+1) + (iz + nbell) * (2*nbell+1) + (ix + nbell)] = exp(-(iz*iz+ix*ix+iy*iy)/s);
	    	}
		}    
    }

	// copy bell coeficients to the GPUs
	float **d_bell = (float**)malloc(ngpu*sizeof(float*));
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_bell[g], (2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float));
		hipMemcpy(d_bell[g], h_bell, (2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float), hipMemcpyDefault);
	}
	/*------------------------------------------------------------*/
	
	
	/*------------------------------------------------------------*/
	/* 3D vector components */
    nc=3;
	ac=sf_maxa(nc  ,0,1);
	/*------------------------------------------------------------*/
	
	
	/*------------------------------------------------------------*/
     /* setup output data files and arrays */
    sf_oaxa(Fdat,ar,1);
    sf_oaxa(Fdat,ac,2);

    sf_setn(at,nt/jdata);
    sf_setd(at,dt*jdata);
    sf_oaxa(Fdat,at,3);

    if(snap) {

		// Used to accumulate wavefield data from other GPUs
		uoz=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
		uox=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
		uoy=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
		h_uoz = (float*)malloc(fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float));
		h_uox = (float*)malloc(fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float));
		h_uoy = (float*)malloc(fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float));

		nqz=sf_n(az);
		nqx=sf_n(ax);
		nqy=sf_n(ay);

		oqz=sf_o(az);
		oqx=sf_o(ax);
		oqy=sf_o(ay);

		dqz=sf_d(az);
		dqx=sf_d(ax);
		dqy=sf_d(ay);

		acz = sf_maxa(nqz,oqz,dqz); if (rank==0) sf_raxa(acz);
		acx = sf_maxa(nqx,oqx,dqx); if (rank==0) sf_raxa(acx);
		acy = sf_maxa(nqy,oqy,dqy); if (rank==0) sf_raxa(acy);

		uc=sf_floatalloc3(sf_n(acz),sf_n(acx),sf_n(acy));

		ntsnap=0;
		for(it=0; it<nt; it++) {
		    if(it%jsnap==0) ntsnap++;
		}
		sf_setn(at,  ntsnap);
		sf_setd(at,dt*jsnap);
		if(verb && rank==0) sf_raxa(at);

		sf_oaxa(Fwfl,acz,1);
		sf_oaxa(Fwfl,acx,2);
		sf_oaxa(Fwfl,acy,3);
		sf_oaxa(Fwfl,ac, 4);
		sf_oaxa(Fwfl,at, 5);
    }
    /*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
    /* read source wavelet(s) and copy to each GPU (into d_ww) */
    
	float *h_ww;
	float **d_ww = (float**)malloc(ngpu*sizeof(float*));

	if (wavSrc){
		ww=sf_floatalloc3(ns,nc,nt); 
	    sf_floatread(ww[0][0],nt*nc*ns,Fwav);

		h_ww = (float*)malloc(ns*nc*nt*sizeof(float));
		for (int t = 0; t < nt; t++){
			for (int c = 0; c < nc; c++){
				for (int s = 0; s < ns; s++){
					h_ww[t * nc * ns + c * ns + s]=ww[t][c][s];
				}
			}
		}

		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			hipMalloc(&d_ww[g], ns*nc*nt*sizeof(float));
			hipMemcpy(d_ww[g], h_ww, ns*nc*nt*sizeof(float), hipMemcpyDefault);
		}
	}
    /*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
	/* data array */
	float *h_dd = (float*)malloc(nr * nc * sizeof(float));
	
	float *h_dd_combined = (float*)malloc(nr * nc * sizeof(float));		
	float *h_dd_locals = (float*)malloc(nr * nc * sizeof(float));
	float *h_dd_allNodes = (float*)malloc(nr * nc * sizeof(float));
	
	float **d_dd = (float**)malloc(ngpu*sizeof(float*));
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_dd[g], nr*nc*sizeof(float));
	}
    /*------------------------------------------------------------*/


	/*------------------------------------------------------------*/
    /* setup source/receiver coordinates */
    ss = (pt3d*) sf_alloc(ns,sizeof(*ss)); 
    rr = (pt3d*) sf_alloc(nr,sizeof(*rr)); 

    pt3dread1(Fsou,ss,ns,3); /* read (x,y,z) coordinates */
    pt3dread1(Frec,rr,nr,3); /* read (x,y,z) coordinates */

	/* calculate 3d linear interpolation coefficients for source locations and copy to each GPU*/
    cs = lint3d_make(ns,ss,fdm);
	float **d_Sw000 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw001 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw010 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw011 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw100 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw101 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw110 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Sw111 = (float**)malloc(ngpu*sizeof(float*));

	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_Sw000[g], ns * sizeof(float));
		hipMalloc(&d_Sw001[g], ns * sizeof(float));
		hipMalloc(&d_Sw010[g], ns * sizeof(float));
		hipMalloc(&d_Sw011[g], ns * sizeof(float));
		hipMalloc(&d_Sw100[g], ns * sizeof(float));
		hipMalloc(&d_Sw101[g], ns * sizeof(float));
		hipMalloc(&d_Sw110[g], ns * sizeof(float));
		hipMalloc(&d_Sw111[g], ns * sizeof(float));
		hipMemcpy(d_Sw000[g], cs->w000, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw001[g], cs->w001, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw010[g], cs->w010, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw011[g], cs->w011, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw100[g], cs->w100, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw101[g], cs->w101, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw110[g], cs->w110, ns * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_Sw111[g], cs->w111, ns * sizeof(float), hipMemcpyDefault);		
	}

	// z, x, and y coordinates of each source
	int **d_Sjz = (int**)malloc(ngpu*sizeof(int*));
	int **d_Sjx = (int**)malloc(ngpu*sizeof(int*));
	int **d_Sjy = (int**)malloc(ngpu*sizeof(int*));
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_Sjz[g], ns * sizeof(int));
		hipMalloc(&d_Sjx[g], ns * sizeof(int));
		hipMalloc(&d_Sjy[g], ns * sizeof(int));
		hipMemcpy(d_Sjz[g], cs->jz, ns * sizeof(int), hipMemcpyDefault);
		hipMemcpy(d_Sjx[g], cs->jx, ns * sizeof(int), hipMemcpyDefault);
		hipMemcpy(d_Sjy[g], cs->jy, ns * sizeof(int), hipMemcpyDefault);	
	}


	/* calculate 3d linear interpolation coefficients for receiver locations and copy to each GPU*/
	cr = lint3d_make(nr,rr,fdm);
	float **d_Rw000 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw001 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw010 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw011 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw100 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw101 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw110 = (float**)malloc(ngpu*sizeof(float*));
	float **d_Rw111 = (float**)malloc(ngpu*sizeof(float*));
	if (interp){
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			hipMalloc(&d_Rw000[g], nr * sizeof(float));
			hipMalloc(&d_Rw001[g], nr * sizeof(float));
			hipMalloc(&d_Rw010[g], nr * sizeof(float));
			hipMalloc(&d_Rw011[g], nr * sizeof(float));
			hipMalloc(&d_Rw100[g], nr * sizeof(float));
			hipMalloc(&d_Rw101[g], nr * sizeof(float));
			hipMalloc(&d_Rw110[g], nr * sizeof(float));
			hipMalloc(&d_Rw111[g], nr * sizeof(float));
			hipMemcpy(d_Rw000[g], cr->w000, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw001[g], cr->w001, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw010[g], cr->w010, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw011[g], cr->w011, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw100[g], cr->w100, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw101[g], cr->w101, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw110[g], cr->w110, nr * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_Rw111[g], cr->w111, nr * sizeof(float), hipMemcpyDefault);
		}
	}

	// z, x, and y coordinates of each receiver
	int **d_Rjz = (int**)malloc(ngpu*sizeof(int*));
	int **d_Rjx = (int**)malloc(ngpu*sizeof(int*));
	int **d_Rjy = (int**)malloc(ngpu*sizeof(int*));
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_Rjz[g], nr * sizeof(int));
		hipMalloc(&d_Rjx[g], nr * sizeof(int));
		hipMalloc(&d_Rjy[g], nr * sizeof(int));
		hipMemcpy(d_Rjz[g], cr->jz, nr * sizeof(int), hipMemcpyDefault);
		hipMemcpy(d_Rjx[g], cr->jx, nr * sizeof(int), hipMemcpyDefault);
		hipMemcpy(d_Rjy[g], cr->jy, nr * sizeof(int), hipMemcpyDefault);	
	}
	/*------------------------------------------------------------*/
	
	
	/*------------------------------------------------------------*/	
    /* setup FD coefficients */
    idz = 1/dz;
    idx = 1/dx;
    idy = 1/dy;
	/*------------------------------------------------------------*/
	
	
	/*------------------------------------------------------------*/ 
	/* read in model density and stiffness arrays */
	
	float *h_ro, *h_c11, *h_c22, *h_c33, *h_c44, *h_c55, *h_c66, *h_c12, *h_c13, *h_c23;
	
	if (nNode == 1){
		float *tt = (float*)malloc(nz*nx*ny*sizeof(float));

	    /* input density */
	    h_ro = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
	    sf_floatread(tt,nz*nx*ny,Fden);     expand_all(tt, h_ro, fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);

	    /* stiffness */
	    h_c11 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
	    h_c22 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c33 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c44 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c55 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c66 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c12 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c13 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float)); 
	    h_c23 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c11,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c22,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);    
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c33,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);    
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c44,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c55,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);    
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c66,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c12,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c13,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    sf_floatread(tt,nz*nx*ny,Fccc);    expand_all(tt,h_c23,fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
	    free(tt);
	}
	else {

		if (rank == 0){
			float *tt = (float*)malloc(((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz*sizeof(float));

		    /* input density */
		    h_ro = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		    sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fden);	expand_low(tt, h_ro, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		
		    /* stiffness */
		    h_c11 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		    h_c22 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c33 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c44 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c55 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c66 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c12 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c13 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c23 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		
		    sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c11, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
			sf_seek(Fccc, 1*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c22, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
			sf_seek(Fccc, 2*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c33, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
			sf_seek(Fccc, 3*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c44, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
			sf_seek(Fccc, 4*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c55, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, 5*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c66, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, 6*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c12, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, 7*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c13, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, 8*ny*nx*nz*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);    expand_low(tt, h_c23, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
			free(tt);
		}
		else if (rank = nNode-1){

			float *tt = (float*)malloc(((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz*sizeof(float));

		    /* input density */
		    h_ro = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));

			sf_seek(Fden, ((nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0); 
		    sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fden);	expand_high(tt, h_ro, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);

			/* stiffness */
		    h_c11 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		    h_c22 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c33 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c44 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c55 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c66 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c12 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c13 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c23 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		
		    sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c11, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (1*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c22, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (2*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c33, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (3*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c44, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (4*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c55, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (5*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c66, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (6*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c12, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (7*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c13, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (8*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,((nyinterior - nb) + (nyinterior * (ngpu-1)))*nx*nz,Fccc);     expand_high(tt, h_c23, fdm->nb, nx, fdm->nxpad, (nyinterior - nb) + (nyinterior * (ngpu-1)), (nyinterior * ngpu), nz, fdm->nzpad);
			free(tt);
		}
		else {
			
			float *tt = (float*)malloc((nyinterior * ngpu)*nx*nz*sizeof(float));

		    /* input density */
		    h_ro = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));

			sf_seek(Fden, ((nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0); 
		    sf_floatread(tt,(nyinterior * ngpu)*nx*nz,Fden);	expand_interior(tt, h_ro, fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);

			/* stiffness */
		    h_c11 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
		    h_c22 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c33 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c44 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c55 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c66 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c12 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c13 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float)); 
		    h_c23 = (float*)malloc(fdm->nzpad * fdm->nxpad * (nyinterior * ngpu) * sizeof(float));
			
			sf_seek(Fccc, (0*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
		    sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c11,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (1*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c22,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (2*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c33,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (3*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c44,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (4*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c55,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);    
		    sf_seek(Fccc, (5*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c66,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (6*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c12,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (7*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c13,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    sf_seek(Fccc, (8*ny*nx*nz + (nyinterior * ngpu * (rank-1) + (nyinterior - nb) + (nyinterior * (ngpu-1))) * nx * nz)*sizeof(float), 0);
			sf_floatread(tt,nz*nx*(nyinterior * ngpu),Fccc);    expand_interior(tt,h_c23,fdm->nb, nx, fdm->nxpad, (nyinterior * ngpu), (nyinterior * ngpu), nz, fdm->nzpad);
		    free(tt);
		}
	}
	
	// device arrays for local GPUs
	float **d_ro = (float**)malloc(ngpu*sizeof(float*));
	float **d_c11 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c22 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c33 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c44 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c55 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c66 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c12 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c13 = (float**)malloc(ngpu*sizeof(float*));
	float **d_c23 = (float**)malloc(ngpu*sizeof(float*));

	// allocate density and stiffness sub-domain arrays on each GPU and copy the data
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		hipMalloc(&d_ro[g] , nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c11[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c22[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c33[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c44[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c55[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c66[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c12[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c13[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_c23[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMemcpy(d_ro[g] , h_ro  + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c11[g], h_c11 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c22[g], h_c22 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c33[g], h_c33 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c44[g], h_c44 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c55[g], h_c55 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c66[g], h_c66 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c12[g], h_c12 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c13[g], h_c13 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
		hipMemcpy(d_c23[g], h_c23 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);	
	}
	
	
	
	/*------------------------------------------------------------*/
	/* Boundary condition setup */
	
	float spo;
	
	float **d_bzl_s = (float**)malloc(ngpu*sizeof(float*));
	float **d_bzh_s = (float**)malloc(ngpu*sizeof(float*));
	float **d_bxl_s = (float**)malloc(ngpu*sizeof(float*));
	float **d_bxh_s = (float**)malloc(ngpu*sizeof(float*));
	float **d_byl_s = (float**)malloc(ngpu*sizeof(float*));
	float **d_byh_s = (float**)malloc(ngpu*sizeof(float*));
	
	if (dabc){
		
		/* ABC set up */
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			hipMalloc(&d_bzl_s[g], fdm->nxpad * nyinterior * sizeof(float));
			hipMalloc(&d_bzh_s[g], fdm->nxpad * nyinterior * sizeof(float));
			hipMalloc(&d_bxl_s[g], fdm->nzpad * nyinterior * sizeof(float));
			hipMalloc(&d_bxh_s[g], fdm->nzpad * nyinterior * sizeof(float));
		}
		
		if (rank==0){
			hipSetDevice(0);
			hipMalloc(&d_byl_s[0], fdm->nzpad * fdm->nxpad * sizeof(float));
		}
		if (rank == nNode-1){
			hipSetDevice(ngpu-1);
			hipMalloc(&d_byh_s[ngpu-1], fdm->nzpad * fdm->nxpad * sizeof(float));
		}
		
		
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid_init_XY(ceil(fdm->nxpad/32.0f), ceil(nyinterior/32.0f));
			dim3 dimBlock_init_XY(32,32);
			initABC_XY<<<dimGrid_init_XY, dimBlock_init_XY>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_bzl_s[g], d_bzh_s[g], d_c55[g], d_ro[g], dt, fdm->dz);
			
			dim3 dimGrid_init_ZY(ceil(fdm->nzpad/32.0f), ceil(nyinterior/32.0f));
			dim3 dimBlock_init_ZY(32,32);
			initABC_ZY<<<dimGrid_init_ZY, dimBlock_init_ZY>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_bxl_s[g], d_bxh_s[g], d_c55[g], d_ro[g], dt, fdm->dx);
		}
		
		if (rank==0){
			hipSetDevice(0);
			dim3 dimGrid_init_XZ(ceil(fdm->nxpad/32.0f), ceil(fdm->nzpad/32.0f));
			dim3 dimBlock_init_XZ(32,32);
			initABC_XZ_low<<<dimGrid_init_XZ, dimBlock_init_XZ>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_byl_s[0], d_c55[0], d_ro[0], dt, fdm->dy);
		}
		if (rank == nNode-1){
			hipSetDevice(ngpu-1);
			dim3 dimGrid_init_XZ(ceil(fdm->nxpad/32.0f), ceil(fdm->nzpad/32.0f));
			dim3 dimBlock_init_XZ(32,32);
			initABC_XZ_high<<<dimGrid_init_XZ, dimBlock_init_XZ>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_byh_s[ngpu-1], d_c55[ngpu-1], d_ro[ngpu-1], dt, fdm->dy);
		}
		
		
		/* sponge set up */
		// sponge coefficients are calculated inside the sponge kernel on GPU based on spo
		spo = (sqrt(2.0) * 4.0f * nb);
	}
	
	/*------------------------------------------------------------*/
	
	
	
	/*------------------------------------------------------------*/
    /* displacement: um = U @ t-1; uo = U @ t; up = U @ t+1 */
	float **d_umx = (float **)malloc(ngpu*sizeof(float*));
	float **d_uox = (float **)malloc(ngpu*sizeof(float*));
	float **d_upx = (float **)malloc(ngpu*sizeof(float*));
	float **d_uax = (float **)malloc(ngpu*sizeof(float*));
	float **d_utx = (float **)malloc(ngpu*sizeof(float*));

	float **d_umy = (float **)malloc(ngpu*sizeof(float*));
	float **d_uoy = (float **)malloc(ngpu*sizeof(float*));
	float **d_upy = (float **)malloc(ngpu*sizeof(float*));
	float **d_uay = (float **)malloc(ngpu*sizeof(float*));
	float **d_uty = (float **)malloc(ngpu*sizeof(float*));

	float **d_umz = (float **)malloc(ngpu*sizeof(float*));
	float **d_uoz = (float **)malloc(ngpu*sizeof(float*));
	float **d_upz = (float **)malloc(ngpu*sizeof(float*));
	float **d_uaz = (float **)malloc(ngpu*sizeof(float*));
	float **d_utz = (float **)malloc(ngpu*sizeof(float*));
	
	float **d_tzz = (float **)malloc(ngpu*sizeof(float*));
	float **d_txx = (float **)malloc(ngpu*sizeof(float*));
	float **d_tyy = (float **)malloc(ngpu*sizeof(float*));
	float **d_txy = (float **)malloc(ngpu*sizeof(float*));
	float **d_tyz = (float **)malloc(ngpu*sizeof(float*));
	float **d_tzx = (float **)malloc(ngpu*sizeof(float*));
	
	float *h_uTemp = (float *)malloc(nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float));
	

	// allocate and initialize displacement, accel, and stress/strain arrasys to 0 on each GPU
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		
		hipMalloc(&d_umx[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_uox[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_upx[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));	
		hipMalloc(&d_uax[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMalloc(&d_umy[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_uoy[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_upy[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));	
		hipMalloc(&d_uay[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMalloc(&d_umz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_uoz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_upz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));	
		hipMalloc(&d_uaz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMalloc(&d_tzz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_tyy[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_txx[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_txy[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_tyz[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMalloc(&d_tzx[g], nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		sf_check_gpu_error(rank, "allocate grid arrays");
		
		hipMemset(d_upz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_uaz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMemset(d_upx[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_uax[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		hipMemset(d_upy[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_uay[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		if (wavSrc){
			hipMemset(d_umz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
			hipMemset(d_uoz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
			
			hipMemset(d_umx[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
			hipMemset(d_uox[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));

			hipMemset(d_umy[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
			hipMemset(d_uoy[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		}
		else {
			// get umz interior chunk
			sf_seek(Fum, (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_umz[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			
			// get uoz interior chunk
			sf_seek(Fuo, (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_uoz[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
						
			// get umx interior chunk
			sf_seek(Fum, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_umx[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			
			// get uox interior chunk
			sf_seek(Fuo, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_uox[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
						
			// get umy interior chunk
			sf_seek(Fum, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_umy[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			
			// get uoy interior chunk
			sf_seek(Fuo, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
			sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, nyinterior, fdm->nzpad);
			hipMemcpy(d_uoy[g] + 4 * fdm->nxpad * fdm->nzpad, h_uTemp, nyinterior * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
						
			// get high halos
			if (rank != nNode-1 || g != ngpu-1){
				
				// umz
				sf_seek(Fum, (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umz[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				// uoz
				sf_seek(Fuo, (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uoz[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				
				// umx 
				sf_seek(Fum, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umx[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
		
				// uox
				sf_seek(Fuo, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uox[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				
				// umy
				sf_seek(Fum, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umy[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
		
				// uoy
				sf_seek(Fuo, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior + nyinterior)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uoy[g] + (4 + nyinterior) * fdm->nxpad * fdm->nzpad, h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
			}
			
			// get low halos
			if (rank != 0 || g != 0){
				
				// umz
				sf_seek(Fum, (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umz[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				// uoz
				sf_seek(Fuo, (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uoz[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				
				// umx 
				sf_seek(Fum, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umx[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
					
				// uox
				sf_seek(Fuo, ((fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uox[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				
				// umy
				sf_seek(Fum, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fum, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_umy[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
					
				// uoy
				sf_seek(Fuo, (2*(fdm->nypad * fdm->nzpad * fdm->nxpad) + (rank * nyinterior * ngpu + g * nyinterior - 4)*fdm->nxpad*fdm->nzpad)*sizeof(float), 0);
				sf_floatreadtransp(h_uTemp, Fuo, fdm->nxpad, 4, fdm->nzpad);
				hipMemcpy(d_uoy[g], h_uTemp, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
			}
		}

		hipMemset(d_tzz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_tyy[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_txx[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_txy[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_tyz[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		hipMemset(d_tzx[g], 0, nylocal * fdm->nzpad * fdm->nxpad * sizeof(float));
		
		sf_check_gpu_error(rank, "initialize grid arrays");
	}
	
	// Used for exchanging halo regions between remote GPUs
	// TODO: all of these probably aren't really needed, could probably re-use some, not sure tho...
	float *h_tzz_l_send, *h_tzz_l_recv, *h_tzz_h_send, *h_tzz_h_recv;
	float *h_tyy_l_send, *h_tyy_l_recv, *h_tyy_h_send, *h_tyy_h_recv;
	float *h_txx_l_send, *h_txx_l_recv, *h_txx_h_send, *h_txx_h_recv;
	float *h_txy_l_send, *h_txy_l_recv, *h_txy_h_send, *h_txy_h_recv;
	float *h_tyz_l_send, *h_tyz_l_recv, *h_tyz_h_send, *h_tyz_h_recv;
	float *h_tzx_l_send, *h_tzx_l_recv, *h_tzx_h_send, *h_tzx_h_recv;
	
	float *h_upz_l_send, *h_upz_l_recv, *h_upz_h_send, *h_upz_h_recv;
	float *h_upy_l_send, *h_upy_l_recv, *h_upy_h_send, *h_upy_h_recv;
	float *h_upx_l_send, *h_upx_l_recv, *h_upx_h_send, *h_upx_h_recv;
	
	if (nNode > 1){
		h_tzz_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzz_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzz_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzz_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_tyy_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyy_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyy_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyy_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_txx_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txx_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txx_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txx_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_txy_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txy_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txy_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_txy_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_tyz_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyz_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyz_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tyz_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_tzx_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzx_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzx_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_tzx_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_upz_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upz_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upz_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upz_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_upy_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upy_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upy_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upy_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));

		h_upx_l_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upx_l_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upx_h_send = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
		h_upx_h_recv = (float*)malloc(4 * fdm->nxpad * fdm->nzpad * sizeof(float));
	}
	
	
	/*------------------------------------------------------------*/
    /* precompute 1/ro * dt^2 									  */	
	/*------------------------------------------------------------*/
	for (int g = 0; g < ngpu; g++){
		hipSetDevice(g);
		dim3 dimGrid1(ceil(fdm->nxpad/8.0f),ceil(fdm->nzpad/8.0f),ceil(nyinterior/8.0f));
		dim3 dimBlock1(8,8,8);
		computeRo<<<dimGrid1, dimBlock1>>>(d_ro[g], dt, fdm->nxpad, fdm->nzpad, nyinterior);
	}
	sf_check_gpu_error(rank, "computeRo Kernel");
		
	int first;
	int last;
	
	/*------------------------------------------------------------*/
    /* 
     *  MAIN LOOP
     */
    /*------------------------------------------------------------*/
    if(verb && rank==0) fprintf(stderr,"\n");
    for (it=0; it<nt; it++) {
		if(verb && rank==0) fprintf(stderr,"\b\b\b\b\b%d",it);
		
		/*------------------------------------------------------------*/
		/* from displacement to strain     OUTSIDE                    */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid2((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
			dim3 dimBlock2(24,24,1);
			if (rank == 0 && g == 0){
				first = 1;
				last = 0;
			}
			else if (rank == nNode-1 && g == ngpu-1){
				first = 0;
				last = 1;
			}
			else {
				first = 0;
				last = 0;
			}
			
			// NOTE: These all have to be seperate kernels because nvcc does something REALLY stupid with register usage when you add if-statements......
			if (!first){	// do the low region
				dispToStrainOutside<<<dimGrid2, dimBlock2, 32*32*3*sizeof(float)>>>(0, fdm->nxpad, nylocal, fdm->nzpad, d_uox[g], d_uoy[g], d_uoz[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g], idx, idy, idz);
			}
			if (!last){		// do the high region
				dispToStrainOutside<<<dimGrid2, dimBlock2, 32*32*3*sizeof(float)>>>(1, fdm->nxpad, nylocal, fdm->nzpad, d_uox[g], d_uoy[g], d_uoz[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g], idx, idy, idz);			
			}
		}
		
		sf_check_gpu_error(rank, "dispToStrain Kernel");
		
		
		/*------------------------------------------------------------*/
		/* from strain to stress        OUTSIDE                       */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid3_outside(ceil(fdm->nxpad/192.0f), fdm->nzpad, 8);
			dim3 dimBlock3_outside(192,1,1);
			strainToStressOutside<<<dimGrid3_outside, dimBlock3_outside>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_c11[g], d_c12[g], d_c13[g], d_c22[g], d_c23[g], d_c33[g], d_c44[g], d_c55[g], d_c66[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g]);
		}
		sf_check_gpu_error(rank, "strainToStress Kernel");
		
			
		/*------------------------------------------------------------*/
		/* free surface       OUTSIDE                                 */
		/*------------------------------------------------------------*/
		if(fsrf) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid4(ceil(fdm->nxpad/8.0f), ceil(fdm->nb/8.0f), 1);
				dim3 dimBlock4(8,8,8);
				freeSurfOutside<<<dimGrid4, dimBlock4>>>(fdm->nxpad, nyinterior, fdm->nzpad, fdm->nb, d_tzz[g], d_tyz[g], d_tzx[g]);
			}
			sf_check_gpu_error(rank, "freeSurf Kernel");
		}
		
		
		/*------------------------------------------------------------*/
		/* inject stress source           OUTSIDE                     */
		/*------------------------------------------------------------*/
		if(ssou && wavSrc) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid5(ns, 1, 1);
				dim3 dimBlock5(2 * nbell + 1, 2 * nbell + 1, 1);
				lint3d_bell_gpuOutside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tzz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuOutside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_txx[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuOutside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tyy[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);					
			}
			sf_check_gpu_error(rank, "lint3d_bell_gpu Kernel");
		}
		
		/*------------------------------------------------------------*/
		/* send halo regions of d_t arrays to remote GPU	          */
		/*------------------------------------------------------------*/
		if (nNode > 1){	// using multiple nodes, must exchange halo regions between neighboring exterior GPUs
			if (rank == 0){
				// get high halo region from d_t array on GPU ngpu-1 and send to rank+1
				hipMemcpy(h_tzz_h_send, d_tzz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyy_h_send, d_tyy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txx_h_send, d_txx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txy_h_send, d_txy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 3, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyz_h_send, d_tyz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 4, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tzx_h_send, d_tzx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 5, MPI_COMM_WORLD, &request);

				sf_check_gpu_error(rank, "send get high");

			}
			else if (rank == nNode-1){
				// get low halo region from d_t arrays on GPU 0 and send to rank-1
				hipMemcpy(h_tzz_l_send, d_tzz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyy_l_send, d_tyy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txx_l_send, d_txx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txy_l_send, d_txy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 3, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyz_l_send, d_tyz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 4, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tzx_l_send, d_tzx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 5, MPI_COMM_WORLD, &request);

				sf_check_gpu_error(rank, "send get low");
				
			}
			else {
				// get low halo region from d_t arrays on GPU 0 and send to rank-1
				hipMemcpy(h_tzz_l_send, d_tzz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyy_l_send, d_tyy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txx_l_send, d_txx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txy_l_send, d_txy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 3, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyz_l_send, d_tyz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 4, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tzx_l_send, d_tzx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 5, MPI_COMM_WORLD, &request);

				// get high halo region from d_t arrays on GPU ngpu-1 and send to rank+1
				hipMemcpy(h_tzz_h_send, d_tzz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyy_h_send, d_tyy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txx_h_send, d_txx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &request);
				hipMemcpy(h_txy_h_send, d_txy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_txy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 3, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tyz_h_send, d_tyz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tyz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 4, MPI_COMM_WORLD, &request);
				hipMemcpy(h_tzx_h_send, d_tzx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_tzx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 5, MPI_COMM_WORLD, &request);

			}

			sf_check_gpu_error(rank, "send remote halos");
		}
		
		/*------------------------------------------------------------*/
		/* start recv'ing halo regions of d_t arrays from remote GPU  */
		/*------------------------------------------------------------*/
		if (nNode > 1){
			if (rank == 0){
				
				// receive low halo region of d_t arrays from rank+1 and copy into GPU ngpu-1 high halo
				MPI_Irecv(h_tzz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_tyy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_txx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &recvRequests[2]);
				MPI_Irecv(h_txy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 3, MPI_COMM_WORLD, &recvRequests[3]);
				MPI_Irecv(h_tyz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 4, MPI_COMM_WORLD, &recvRequests[4]);
				MPI_Irecv(h_tzx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 5, MPI_COMM_WORLD, &recvRequests[5]);
				
			}
			else if (rank == nNode - 1){
				
				// receive high halo region of d_t arrays from rank-1 and copy into GPU 0 low halo
				MPI_Irecv(h_tzz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_tyy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_txx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &recvRequests[2]);
				MPI_Irecv(h_txy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 3, MPI_COMM_WORLD, &recvRequests[3]);
				MPI_Irecv(h_tyz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 4, MPI_COMM_WORLD, &recvRequests[4]);
				MPI_Irecv(h_tzx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 5, MPI_COMM_WORLD, &recvRequests[5]);
				
			}
			
			else {
				
				// receive high halo region of d_t arrays from from rank-1 and copy to GPU 0
				MPI_Irecv(h_tzz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_tyy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_txx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &recvRequests[2]);
				MPI_Irecv(h_txy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 3, MPI_COMM_WORLD, &recvRequests[3]);
				MPI_Irecv(h_tyz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 4, MPI_COMM_WORLD, &recvRequests[4]);
				MPI_Irecv(h_tzx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 5, MPI_COMM_WORLD, &recvRequests[5]);

				// receive low halo region of d_t arrays from from rank+1 and copy to GPU ngpu-1
				MPI_Irecv(h_tzz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &recvRequests[6]);
				MPI_Irecv(h_tyy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &recvRequests[7]);
				MPI_Irecv(h_txx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &recvRequests[8]);
				MPI_Irecv(h_txy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 3, MPI_COMM_WORLD, &recvRequests[9]);
				MPI_Irecv(h_tyz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 4, MPI_COMM_WORLD, &recvRequests[10]);
				MPI_Irecv(h_tzx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 5, MPI_COMM_WORLD, &recvRequests[11]);
								
			}
			
		}
		
		
		/*------------------------------------------------------------*/
		/* exchange halo regions of d_t arrays between local GPUs     */
		/*------------------------------------------------------------*/		
		if (ngpu > 1){ // using multiple GPUs, must exchange halo regions between neighboring GPUs
			// high halo region of d_t arrays on GPU 0 to GPU 1
			hipMemcpy(d_tzz[1], d_tzz[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tyy[1], d_tyy[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_txx[1], d_txx[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_txy[1], d_txy[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tyz[1], d_tyz[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tzx[1], d_tzx[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			
			// exchange halo regions of d_t arrays between all internal GPUs
			for (int g = 1; g < ngpu-1; g++){
				// high halo region of GPU g to low halo region of GPU g+1
				hipMemcpy(d_tzz[g+1], d_tzz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[g+1], d_tyy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[g+1], d_txx[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[g+1], d_txy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[g+1], d_tyz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[g+1], d_tzx[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				// low halo region of GPU g to high halo region of GPU g-1
				hipMemcpy(d_tzz[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tzz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tyy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_txx[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_txy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tyz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[g-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tzx[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			}
			
			// low halo region of d_t arrays on GPU (ngpu-1) to GPU (ngpu-2)
			hipMemcpy(d_tzz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tzz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tyy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tyy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_txx[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_txx[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_txy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_txy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tyz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tyz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_tzx[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_tzx[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
		
			sf_check_gpu_error(rank, "exchange local halos");
		}
		
		
		/*------------------------------------------------------------*/
		/* from displacement to strain       INSIDE                   */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid2((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
			dim3 dimBlock2(24,24,1);
			dispToStrainInside<<<dimGrid2, dimBlock2, 32*32*3*sizeof(float)>>>(fdm->nxpad, nylocal, fdm->nzpad, d_uox[g], d_uoy[g], d_uoz[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g], idx, idy, idz);						
		}
		sf_check_gpu_error(rank, "dispToStrain Kernel");
		
		
		/*------------------------------------------------------------*/
		/* from strain to stress     INSIDE                           */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid3_inside(ceil(fdm->nxpad/192.0f), fdm->nzpad, nyinterior-8);
			dim3 dimBlock3_inside(192,1,1);
			strainToStressInside<<<dimGrid3_inside, dimBlock3_inside>>>(fdm->nxpad, fdm->nzpad, nyinterior, d_c11[g], d_c12[g], d_c13[g], d_c22[g], d_c23[g], d_c33[g], d_c44[g], d_c55[g], d_c66[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g]);
		}
		sf_check_gpu_error(rank, "strainToStress Kernel");
		
			
		/*------------------------------------------------------------*/
		/* free surface         INSIDE                                */
		/*------------------------------------------------------------*/
		if(fsrf) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid4(ceil(fdm->nxpad/8.0f), ceil(fdm->nb/8.0f), ceil(nyinterior/8.0f));
				dim3 dimBlock4(8,8,8);
				freeSurfInside<<<dimGrid4, dimBlock4>>>(fdm->nxpad, nyinterior, fdm->nzpad, fdm->nb, d_tzz[g], d_tyz[g], d_tzx[g]);
				
			}
			sf_check_gpu_error(rank, "freeSurf Kernel");
		}
		
		
		/*------------------------------------------------------------*/
		/* inject stress source     INSIDE                            */
		/*------------------------------------------------------------*/
		if(ssou && wavSrc) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid5(ns, 1, 1);
				dim3 dimBlock5(2 * nbell + 1, 2 * nbell + 1, 1);
				lint3d_bell_gpuInside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tzz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuInside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_txx[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuInside<<<dimGrid5, dimBlock5>>>(rank, ngpu, g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tyy[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);	
			}
			sf_check_gpu_error(rank, "lint3d_bell_gpu Kernel");
		}
		
		
		/*------------------------------------------------------------*/
		/* copy halos from d_t arrays from remote GPU into local GPU  */
		/*------------------------------------------------------------*/
		// Wait for all MPI transfers to complete
		if (nNode > 1){
			if (rank == 0 || rank == nNode-1){
				MPI_Waitall(6, recvRequests, recvStatuses);
			}
			else {
				MPI_Waitall(12, recvRequests, recvStatuses);
			}
		}
		if (nNode > 1){
			if (rank == 0){
				
				hipMemcpy(d_tzz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tzz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tyy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_txx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_txy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tyz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tzx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

				sf_check_gpu_error(rank, "exchange insert low");
			}
			else if (rank == nNode - 1){
				
				hipMemcpy(d_tzz[0], h_tzz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[0], h_tyy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[0], h_txx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[0], h_txy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[0], h_tyz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[0], h_tzx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				sf_check_gpu_error(rank, "exchange insert high");
			}
			else {
				
				hipMemcpy(d_tzz[0], h_tzz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[0], h_tyy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[0], h_txx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[0], h_txy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[0], h_tyz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[0], h_tzx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				hipMemcpy(d_tzz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tzz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tyy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_txx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_txy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_txy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tyz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tyz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_tzx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_tzx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
			}
		}
		

		
		/*------------------------------------------------------------*/
		/* from stress to acceleration               OUTSIDE          */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid6((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
			dim3 dimBlock6(24,24,1);
			if (rank == 0 && g == 0){
				first = 1;
				last = 0;
			}
			else if (rank == nNode-1 && g == ngpu-1){
				first = 0;
				last = 1;
			}
			else {
				first = 0;
				last = 0;
			}
			if (!first){	// do the low region
				stressToAccelOutside<<<dimGrid6, dimBlock6, 32*32*5*sizeof(float)>>>(0, fdm->nxpad, fdm->nzpad, nylocal, idx, idy, idz, d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tzx[g], d_tyz[g], d_uax[g], d_uay[g], d_uaz[g]);
			}
			if (!last){
				stressToAccelOutside<<<dimGrid6, dimBlock6, 32*32*5*sizeof(float)>>>(1, fdm->nxpad, fdm->nzpad, nylocal, idx, idy, idz, d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tzx[g], d_tyz[g], d_uax[g], d_uay[g], d_uaz[g]);
			}
		}
		sf_check_gpu_error(rank, "stressToAccel Kernel");
		
		
		/*------------------------------------------------------------*/
		/* inject acceleration source      OUTSIDE                    */
		/*------------------------------------------------------------*/
		if(!ssou && wavSrc) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid7(ns, 1, 1);
				dim3 dimBlock7(2 * nbell + 1, 2 * nbell + 1, 1);
				lint3d_bell_gpuOutside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uaz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuOutside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uax[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuOutside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uay[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);	
			}
		
			sf_check_gpu_error(rank, "lint3d_bell_gpu Kernel");
		}
		
		
		
		/*------------------------------------------------------------*/
		/* step forward in time        OUTSIDE                        */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid8_outside(ceil(fdm->nxpad/192.0f), fdm->nzpad, 8);
			dim3 dimBlock8_outside(192,1,1);
			stepTimeOutside<<<dimGrid8_outside, dimBlock8_outside>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_ro[g], d_uox[g], d_umx[g], d_uax[g], d_upx[g], d_uoy[g], d_umy[g], d_uay[g], d_upy[g], d_uoz[g], d_umz[g], d_uaz[g], d_upz[g]);
		}
		sf_check_gpu_error(rank, "stepTime Kernel");
		

		/*------------------------------------------------------------*/
		/* apply boundary conditions          OUTSIDE                 */
		/*------------------------------------------------------------*/
		if(dabc){
			
			/* One-way Absorbing BC */
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				
				dim3 dimGrid_abc_XY(ceil(fdm->nxpad/32.0f),2,2);
				dim3 dimBlock_abc_XY(32,4,1);
				abcone3d_apply_XY_outside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upx[g], d_uox[g], d_bzl_s[g], d_bzh_s[g]);
				abcone3d_apply_XY_outside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upy[g], d_uoy[g], d_bzl_s[g], d_bzh_s[g]);
				abcone3d_apply_XY_outside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upz[g], d_uoz[g], d_bzl_s[g], d_bzh_s[g]);
				
				dim3 dimGrid_abc_ZY(2, 2, ceil(fdm->nzpad/32.0f));
				dim3 dimBlock_abc_ZY(1,4,32);
				abcone3d_apply_ZY_outside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upx[g], d_uox[g], d_bxl_s[g], d_bxh_s[g]);
				abcone3d_apply_ZY_outside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upy[g], d_uoy[g], d_bxl_s[g], d_bxh_s[g]);
				abcone3d_apply_ZY_outside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upz[g], d_uoz[g], d_bxl_s[g], d_bxh_s[g]);
			}
			
			
			/* sponge BC */
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid_spng_XY(ceil(fdm->nxpad/192.0f),4,2);
				dim3 dimBlock_spng_XY(192,1,1);                                            
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			                                                                    
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			                                                                    
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_outside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			
			
				dim3 dimGrid_spng_ZY(ceil(nb/8.0f),ceil(fdm->nzpad/8.0f),2);
				dim3 dimBlock_spng_ZY(8,8,4);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
							                                                                    
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
							                                                                    
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_outside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
			}
			
			sf_check_gpu_error(rank, "Boundary Condition Kernels");
			
		}

		

		/*------------------------------------------------------------*/
		/* start sending halo regions of d_up arrays to remote GPUs   */
		/*------------------------------------------------------------*/
		if (nNode > 1){	// using multiple nodes, must exchange halo regions between neighboring nodes GPUs
			if (rank == 0){
				// get high halo region from d_t array on GPU ngpu-1 and send to rank+1
				hipMemcpy(h_upx_h_send, d_upx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upy_h_send, d_upy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upz_h_send, d_upz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &request);
				
			}
			else if (rank == nNode-1){
				// get low halo region from d_t arrays on GPU 0 and send to rank-1
				hipMemcpy(h_upx_l_send, d_upx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upy_l_send, d_upy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upz_l_send, d_upz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &request);
				
			}
			else {
				// get low halo region from d_t arrays on GPU 0 and send to rank-1
				hipMemcpy(h_upx_l_send, d_upx[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upx_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upy_l_send, d_upy[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upy_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upz_l_send, d_upz[0] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upz_l_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &request);
				
				// get high halo region from d_t arrays on GPU ngpu-1 and send to rank+1
				hipMemcpy(h_upx_h_send, d_upx[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upx_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upy_h_send, d_upy[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upy_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &request);
				hipMemcpy(h_upz_h_send, d_upz[ngpu-1] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				MPI_Isend(h_upz_h_send, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &request);
			}
		}
		
		
		/*------------------------------------------------------------*/
		/* start recv'ing halo regions of d_up arrays from remote GPU */
		/*------------------------------------------------------------*/
		if (nNode > 1){	// using multiple nodes, must exchange halo regions between neighboring nodes GPUs
			if (rank == 0){
				// receive low halo region of d_t arrays from from rank+1 and copy to GPU ngpu-1
				MPI_Irecv(h_upx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_upy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_upz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &recvRequests[2]);
			}
			else if (rank == nNode-1){				
				// receive high halo region of d_t arrays from from rank-1 and copy to GPU 0
				MPI_Irecv(h_upx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_upy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_upz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &recvRequests[2]);
			}
			else {				
				// receive high halo region of d_t arrays from from rank-1 and copy to GPU 0
				MPI_Irecv(h_upx_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 0, MPI_COMM_WORLD, &recvRequests[0]);
				MPI_Irecv(h_upy_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 1, MPI_COMM_WORLD, &recvRequests[1]);
				MPI_Irecv(h_upz_h_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank-1, 2, MPI_COMM_WORLD, &recvRequests[2]);
		
				// receive low halo region of d_t arrays from from rank+1 and copy to GPU ngpu-1
				MPI_Irecv(h_upx_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 0, MPI_COMM_WORLD, &recvRequests[3]);
				MPI_Irecv(h_upy_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 1, MPI_COMM_WORLD, &recvRequests[4]);
				MPI_Irecv(h_upz_l_recv, 4 * fdm->nxpad * fdm->nzpad, MPI_FLOAT, rank+1, 2, MPI_COMM_WORLD, &recvRequests[5]);
			}
		}
		
		
		/*------------------------------------------------------------*/
		/* exchange halo regions of d_up arrays between local GPUs    */
		/*------------------------------------------------------------*/
		if (ngpu > 1){ // using multiple local GPUs, must exchange halo regions between neighboring local GPUs
			// high halo region of d_up arrays on GPU 0 to GPU 1
			hipMemcpy(d_upx[1], d_upx[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_upy[1], d_upy[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_upz[1], d_upz[0] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			
			// exchange halo regions of d_up arrays between all internal GPUs
			for (int g = 1; g < ngpu-1; g++){
				// high halo region of GPU g to low halo region of GPU g+1
				hipMemcpy(d_upx[g+1], d_upx[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[g+1], d_upy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[g+1], d_upz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				// low halo region of GPU g to high halo region of GPU g-1
				hipMemcpy(d_upx[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal - 4)), d_upx[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal - 4)), d_upy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal - 4)), d_upz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			}
			
			// low halo region of d_up arrays on GPU (ngpu-1) to GPU (ngpu-2)
			hipMemcpy(d_upx[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_upx[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_upy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_upy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			hipMemcpy(d_upz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), d_upz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
		}
		
		
		/*------------------------------------------------------------*/
		/* from stress to acceleration         INSIDE                 */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid6((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
			dim3 dimBlock6(24,24,1);
			stressToAccelInside<<<dimGrid6, dimBlock6, 32*32*5*sizeof(float)>>>(fdm->nxpad, fdm->nzpad, nylocal, idx, idy, idz, d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tzx[g], d_tyz[g], d_uax[g], d_uay[g], d_uaz[g]);
		}
		sf_check_gpu_error(rank, "stressToAccel Kernel");
		
		
		/*------------------------------------------------------------*/
		/* inject acceleration source      INSIDE                     */
		/*------------------------------------------------------------*/
		if(!ssou && wavSrc) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid7(ns, 1, 1);
				dim3 dimBlock7(2 * nbell + 1, 2 * nbell + 1, 1);
				lint3d_bell_gpuInside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uaz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuInside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uax[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
				lint3d_bell_gpuInside<<<dimGrid7, dimBlock7>>>(rank, ngpu, g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uay[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);	
			}
			sf_check_gpu_error(rank, "lint3d_bell_gpu Kernel");
		}
		
		
		/*------------------------------------------------------------*/
		/* step forward in time        INSIDE                         */
		/*------------------------------------------------------------*/
		for (int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			dim3 dimGrid8_inside(ceil(fdm->nxpad/192.0f), fdm->nzpad, nyinterior-8);
			dim3 dimBlock8_inside(192,1,1);
			stepTimeInside<<<dimGrid8_inside, dimBlock8_inside>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_ro[g], d_uox[g], d_umx[g], d_uax[g], d_upx[g], d_uoy[g], d_umy[g], d_uay[g], d_upy[g], d_uoz[g], d_umz[g], d_uaz[g], d_upz[g]);
		}
		sf_check_gpu_error(rank, "stepTime Kernel");
		
		
		/*------------------------------------------------------------*/
		/* apply boundary conditions          INSIDE                  */
		/*------------------------------------------------------------*/
		if(dabc){
			
			/* One-way Absorbing BC */
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				
				dim3 dimGrid_abc_XY(ceil(fdm->nxpad/32.0f),ceil((nyinterior - 8)/32.0f),2);
				dim3 dimBlock_abc_XY(32,32,1);
				abcone3d_apply_XY_inside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upx[g], d_uox[g], d_bzl_s[g], d_bzh_s[g]);
				abcone3d_apply_XY_inside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upy[g], d_uoy[g], d_bzl_s[g], d_bzh_s[g]);
				abcone3d_apply_XY_inside<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upz[g], d_uoz[g], d_bzl_s[g], d_bzh_s[g]);
				
				dim3 dimGrid_abc_ZY(2, ceil((nyinterior-8)/32.0f), ceil(fdm->nzpad/32.0f));
				dim3 dimBlock_abc_ZY(1,32,32);
				abcone3d_apply_ZY_inside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upx[g], d_uox[g], d_bxl_s[g], d_bxh_s[g]);
				abcone3d_apply_ZY_inside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upy[g], d_uoy[g], d_bxl_s[g], d_bxh_s[g]);
				abcone3d_apply_ZY_inside<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(fdm->nxpad, nyinterior, fdm->nzpad, d_upz[g], d_uoz[g], d_bxl_s[g], d_bxh_s[g]);
			}
			
			if (rank == 0){
				hipSetDevice(0);
				dim3 dimGrid_abc_XZ(ceil(fdm->nxpad/32.0f),1,ceil(fdm->nzpad/32.0f));
				dim3 dimBlock_abc_XZ(32,1,32);
				abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_upx[0], d_uox[0], d_byl_s[0]);
				abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_upy[0], d_uoy[0], d_byl_s[0]);
				abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_upz[0], d_uoz[0], d_byl_s[0]);
				
			}
			
			if (rank == nNode-1){
				hipSetDevice(ngpu-1);
				dim3 dimGrid_abc_XZ(ceil(fdm->nxpad/32.0f),1,ceil(fdm->nzpad/32.0f));
				dim3 dimBlock_abc_XZ(32,1,32);
				abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal, fdm->nzpad, d_upx[ngpu-1], d_uox[ngpu-1], d_byh_s[ngpu-1]);
				abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal, fdm->nzpad, d_upy[ngpu-1], d_uoy[ngpu-1], d_byh_s[ngpu-1]);
				abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal, fdm->nzpad, d_upz[ngpu-1], d_uoz[ngpu-1], d_byh_s[ngpu-1]);
			}
			
			/* sponge BC */
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				dim3 dimGrid_spng_XY(ceil(fdm->nxpad/192.0f),(nyinterior - 8),1);
				dim3 dimBlock_spng_XY(192,1,1);                                            
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			                                                                   
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			                                                                   
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
				sponge3d_apply_XY_inside<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
			
			
				dim3 dimGrid_spng_ZY(ceil(nb/8.0f),ceil(fdm->nzpad/8.0f),ceil((nyinterior - 8)/8.0f));
				dim3 dimBlock_spng_ZY(8,8,8);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
							                                                                   
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
						                                                                       
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
				sponge3d_apply_ZY_inside<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
			}
			
			if (rank == 0){
				hipSetDevice(0);
				dim3 dimGrid_spng_XZ(ceil(fdm->nxpad/192.0f),1,fdm->nzpad);
				dim3 dimBlock_spng_XZ(192,1,1);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umz[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoz[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upz[0], fdm->nxpad, fdm->nzpad, nb);
					                                                                          
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umx[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uox[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upx[0], fdm->nxpad, fdm->nzpad, nb);
					                                                                          
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umy[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoy[0], fdm->nxpad, fdm->nzpad, nb);
				sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upy[0], fdm->nxpad, fdm->nzpad, nb);
			}
			
			if (rank == nNode-1){
				hipSetDevice(ngpu-1);
				dim3 dimGrid_spng_XZ(ceil(fdm->nxpad/192.0f),1,fdm->nzpad);
				dim3 dimBlock_spng_XZ(192,1,1);                                                                      
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umz[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoz[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upz[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
					                                                                                                               
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umx[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uox[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upx[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
					                                                                                                               
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umy[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoy[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
				sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upy[ngpu-1], fdm->nxpad, nylocal, fdm->nzpad, nb);
			}
			
			sf_check_gpu_error(rank, "Boundary Condition Kernels");
			
		}
		
		
		/*------------------------------------------------------------*/
		/* copy halos from d_up arrays from remote GPU into local GPU */
		/*------------------------------------------------------------*/
		// Wait for all MPI transfers to complete
		if (nNode > 1){
			if (rank == 0 || rank == nNode-1){
				MPI_Waitall(3, recvRequests, recvStatuses);
			}
			else {
				MPI_Waitall(6, recvRequests, recvStatuses);
			}
		}
		if (nNode > 1){	// using multiple nodes, must exchange halo regions between neighboring nodes GPUs
			if (rank == 0){
				hipMemcpy(d_upx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			}
			else if (rank == nNode-1){				
				hipMemcpy(d_upx[0], h_upx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[0], h_upy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[0], h_upz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
		
			}
			else {				
				hipMemcpy(d_upx[0], h_upx_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[0], h_upy_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[0], h_upz_h_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				
				hipMemcpy(d_upx[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upx_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upy[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upy_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
				hipMemcpy(d_upz[ngpu-1] + (fdm->nxpad * fdm->nzpad * (nyinterior + 4)), h_upz_l_recv, 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
			}
		}
		 
					
		/* circulate wavefield arrays */
		for (int g = 0; g < ngpu; g++){
			d_utz[g]=d_umz[g]; d_uty[g]=d_umy[g]; d_utx[g]=d_umx[g];
			d_umz[g]=d_uoz[g]; d_umy[g]=d_uoy[g]; d_umx[g]=d_uox[g];
			d_uoz[g]=d_upz[g]; d_uoy[g]=d_upy[g]; d_uox[g]=d_upx[g];
			d_upz[g]=d_utz[g]; d_upy[g]=d_uty[g]; d_upx[g]=d_utx[g];
		}
		
		
		/*------------------------------------------------------------*/
		/* cut wavefield and save */
		/*------------------------------------------------------------*/
		if(snap && it%jsnap==0) {
			
			if (rank == 0){	// accumulate wavefield data and write to file
				
				// write local GPU's portions of wavefield data into output arrays
				if (nNode == 1 && ngpu == 1){
					hipMemcpy(h_uox, d_uox[0], fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
					hipMemcpy(h_uoy, d_uoy[0], fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
					hipMemcpy(h_uoz, d_uoz[0], fdm->nypad * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
				}
				else {
					for (int g = 0; g < ngpu; g++){
						hipMemcpy(h_uox + g * (nyinterior * fdm->nzpad * fdm->nxpad), d_uox[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
						hipMemcpy(h_uoy + g * (nyinterior * fdm->nzpad * fdm->nxpad), d_uoy[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
						hipMemcpy(h_uoz + g * (nyinterior * fdm->nzpad * fdm->nxpad), d_uoz[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
					}
				}
			
				// receive wavefield from other GPUs and write to output arrays
				for (int r = 1; r < nNode; r++){
					MPI_Recv(h_uox + r * ngpu * nyinterior * fdm->nzpad * fdm->nxpad, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, r, 0, MPI_COMM_WORLD, &status);
					MPI_Recv(h_uoy + r * ngpu * nyinterior * fdm->nzpad * fdm->nxpad, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, r, 1, MPI_COMM_WORLD, &status);
					MPI_Recv(h_uoz + r * ngpu * nyinterior * fdm->nzpad * fdm->nxpad, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, r, 2, MPI_COMM_WORLD, &status);
				}
			
			
				for (int y = 0; y < fdm->nypad; y++){
					for (int z = 0; z < fdm->nzpad; z++){
						for (int x = 0; x < fdm->nxpad; x++){
							uox[y][x][z] = h_uox[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
							uoy[y][x][z] = h_uoy[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
							uoz[y][x][z] = h_uoz[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
						}
					}
				}
				
				// Write complete wavefield to output file
				// sf_floatwrite(h_uoz,fdm->nypad * fdm->nzpad * fdm->nxpad,Fwfl);
				// sf_floatwrite(h_uox,fdm->nypad * fdm->nzpad * fdm->nxpad,Fwfl);
				// sf_floatwrite(h_uoy,fdm->nypad * fdm->nzpad * fdm->nxpad,Fwfl);

				// Write wavefield arrays to output file
				cut3d(uoz,uc,fdm,acz,acx,acy);	// flip axes back
				sf_floatwrite(uc[0][0],sf_n(acx)*sf_n(acy)*sf_n(acz),Fwfl);
				
				cut3d(uox,uc,fdm,acz,acx,acy);	// flip axes back
				sf_floatwrite(uc[0][0],sf_n(acx)*sf_n(acy)*sf_n(acz),Fwfl);
				
				cut3d(uoy,uc,fdm,acz,acx,acy);	// flip axes back
				sf_floatwrite(uc[0][0],sf_n(acx)*sf_n(acy)*sf_n(acz),Fwfl);
			
			}
			else {	// send wavefield data to rank0
			
				// write local GPU's portions of wavefield data into output arrays
				for (int g = 0; g < ngpu; g++){
					hipMemcpy(h_uox + g * nyinterior * fdm->nzpad * fdm->nxpad, d_uox[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
					hipMemcpy(h_uoy + g * nyinterior * fdm->nzpad * fdm->nxpad, d_uoy[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
					hipMemcpy(h_uoz + g * nyinterior * fdm->nzpad * fdm->nxpad, d_uoz[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
				}
			
				MPI_Send(h_uox, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
				MPI_Send(h_uoy, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, 0, 1, MPI_COMM_WORLD);
				MPI_Send(h_uoz, ngpu * nyinterior * fdm->nzpad * fdm->nxpad, MPI_FLOAT, 0, 2, MPI_COMM_WORLD);
			}

		}
		
		/*------------------------------------------------------------*/
		/* extract receiver data									  */
		/*------------------------------------------------------------*/
		if(it%jdata==0) {
			for (int g = 0; g < ngpu; g++){
				hipSetDevice(g);
				hipMemset(d_dd[g], 0, nr*nc*sizeof(float));
				dim3 dimGrid_extract(MIN(nr,ceil(nr/1024.0f)), 1, 1);
				dim3 dimBlock_extract(MIN(nr, 1024), 1, 1);
				if (interp){
					lint3d_extract_gpu<<<dimGrid_extract, dimBlock_extract>>>(rank, ngpu, g, d_dd[g], nr, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_uox[g], d_uoy[g], d_Rjz[g], d_Rjx[g], d_Rjy[g], d_Rw000[g], d_Rw001[g], d_Rw010[g], d_Rw011[g], d_Rw100[g], d_Rw101[g], d_Rw110[g], d_Rw111[g]);
					sf_check_gpu_error(rank, "lint3d_extract kernel");
				}
				else {
					extract_gpu<<<dimGrid_extract, dimBlock_extract>>>(rank, ngpu, g, d_dd[g], nr, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_uox[g], d_uoy[g], d_Rjz[g], d_Rjx[g], d_Rjy[g]);
					sf_check_gpu_error(rank, "extract_gpu kernel");
				}
			}
			
			// copy local GPU 0's receiver data into h_dd_locals
			hipMemcpy(h_dd_locals, d_dd[0], nr * nc * sizeof(float), hipMemcpyDefault);
			
			// add all other GPU's recever data to h_dd_combined
			for (int g = 1; g < ngpu; g++){
				hipMemcpy(h_dd, d_dd[g], nr * nc * sizeof(float), hipMemcpyDefault);
				for (int i = 0; i < nr * nc; i++){
					h_dd_locals[i] += h_dd[i];
				}
			}
			
			// MPI Reduce with SUM on the h_dd_locals arrays from each GPU to accumulate reciever data into h_dd_allNodes
			MPI_Reduce(h_dd_locals, h_dd_allNodes, nr * nc, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD);
			MPI_Barrier(MPI_COMM_WORLD);
			
			if (rank == 0){
				// write receiver data to output file
				sf_floatwrite(h_dd_allNodes, nr*nc, Fdat);
			}
			
		}
		
		
	} // END MAIN LOOP
	
	
	/*------------------------------------------------------------*/
    /* deallocate host arrays */

	if (wavSrc){
		free(**ww); free(*ww); free(ww); free(h_ww);
	}
	free(h_dd); free(h_dd_combined);
	free(ss); free(rr);
	free(h_bell);
	free(h_ro);
	free(h_c11); free(h_c22); free(h_c33); free(h_c44); free(h_c55); free(h_c66); free(h_c12); free(h_c13); free(h_c23);
	
	if (snap){
		free(h_uoz); free(h_uox); free(h_uoy);
		free(**uc);  free(*uc);  free(uc);
		free(**uoz); free(*uoz); free(uoz);
	    free(**uox); free(*uox); free(uox);
	    free(**uoy); free(*uoy); free(uoy);
	}
	
	
	/*------------------------------------------------------------*/
    /* deallocate GPU arrays */

	for (int g = 0; g < ngpu; g++){
	
		if (wavSrc){
			hipFree(&d_ww[g]);
		}
		hipFree(&d_dd[g]);
		hipFree(&d_bell[g]);
	
		hipFree(&d_ro[g]);
		hipFree(&d_c11[g]);
		hipFree(&d_c22[g]);
		hipFree(&d_c33[g]);
		hipFree(&d_c44[g]);
		hipFree(&d_c55[g]);
		hipFree(&d_c66[g]);
		hipFree(&d_c12[g]);
		hipFree(&d_c13[g]);
		hipFree(&d_c23[g]);
	
		if (dabc){
			hipFree(&d_bzl_s[g]);
			hipFree(&d_bzh_s[g]);
			hipFree(&d_bxl_s[g]);
			hipFree(&d_bxh_s[g]);
			hipFree(&d_byl_s[0]);
			hipFree(&d_byh_s[ngpu-1]);
		}
	
		hipFree(&d_umx[g]); hipFree(&d_umy[g]); hipFree(&d_umz[g]);
		hipFree(&d_uox[g]); hipFree(&d_uoy[g]); hipFree(&d_uoz[g]);
		hipFree(&d_upx[g]); hipFree(&d_upy[g]); hipFree(&d_upz[g]);
		hipFree(&d_uax[g]); hipFree(&d_uay[g]); hipFree(&d_uaz[g]);
	
		hipFree(&d_tzz[g]); hipFree(&d_tyy[g]); hipFree(&d_txx[g]); 
		hipFree(&d_txy[g]); hipFree(&d_tyz[g]); hipFree(&d_tzx[g]);
	
		hipFree(&d_Sjz[g]);
		hipFree(&d_Sjx[g]);
		hipFree(&d_Sjy[g]);
		hipFree(&d_Sw000[g]);
		hipFree(&d_Sw001[g]);
		hipFree(&d_Sw010[g]);
		hipFree(&d_Sw011[g]);
		hipFree(&d_Sw100[g]);
		hipFree(&d_Sw101[g]);
		hipFree(&d_Sw110[g]);
		hipFree(&d_Sw111[g]);
	
		hipFree(&d_Rjz[g]);
		hipFree(&d_Rjx[g]);
		hipFree(&d_Rjy[g]);
		if (interp){
			hipFree(&d_Rw000[g]);
			hipFree(&d_Rw001[g]);
			hipFree(&d_Rw010[g]);
			hipFree(&d_Rw011[g]);
			hipFree(&d_Rw100[g]);
			hipFree(&d_Rw101[g]);
			hipFree(&d_Rw110[g]);
			hipFree(&d_Rw111[g]);
		}
	}
	
	
	MPI_Finalize();
	sf_close();
	exit(0);
	
}








